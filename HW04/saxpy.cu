
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);
    float *h_x, *h_y, *d_x, *d_y;

    h_x = (float*)malloc(size);
    h_y = (float*)malloc(size);
    for (int i = 0; i < N; i++){
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0F, d_x, d_y);

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
    printf("y[0] = %f\n", h_y[0]);

    hipFree(d_x);
    hipFree(d_y);
    free(h_x);
    free(h_y);

    return 0;
}